#include "hip/hip_runtime.h"
#include "wsart_support.h"

int TIFFReadContigStripData(TIFF* tif, int elemsize, int bpp, void* dataptr) {
    unsigned char *buf;
    tsize_t scanlinesize = TIFFScanlineSize(tif);
    long tiffstripsize;
    unsigned char* p;
    unsigned short* ps;
    int i,j;
    uint32 row, h, w, ofs;
    uint32 rowsperstrip = (uint32)-1;

	p = (unsigned char*)dataptr; ps= (unsigned short*)dataptr;

	tiffstripsize = TIFFStripSize(tif); /* usually scan line (=row) size * rows per strip */
	buf = (unsigned char *)_TIFFmalloc(tiffstripsize);
	if (buf) 
	{

		TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &h);
		TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &w);
		TIFFGetField(tif, TIFFTAG_ROWSPERSTRIP, &rowsperstrip);

		printf("Image length %d, image width (height) %d\n",(int)h,(int)w);
		printf("Scan line size: %d, strip size %li, rows per strip %d\n",
					(int)scanlinesize,tiffstripsize,rowsperstrip);
		if (bpp==12)
		{
			printf("Performing expansion from 12 to 16 bits/sample\n");
		}
		else if (bpp==1)	/* 1 bit per pixel (fax) - expand */
		{
			printf("Performing expansion from 1 to 8 bits/sample\n");
		}
		else if ((bpp==16)  && (elemsize==6))
		{
			printf("Performing conversion from 48-bit RGB to Z stack\n");
		}
		

		ofs=0;
		for (row = 0; row < h; row += rowsperstrip) 
		{
			/* nrow is the number of rows to be read. At the end of the image we have less of them */
			uint32 nrows = (row+rowsperstrip > h ? h-row : rowsperstrip);

			/* Read the next strip, decode compression if necessary */
			tstrip_t strip = TIFFComputeStrip(tif, row, 0);
			if (TIFFReadEncodedStrip(tif, strip, buf, nrows*scanlinesize) < 0) 
			{
				printf("Error reading strip data");
				return -1;
			} 
		
			if (bpp==12)		/* 12 bit per pixel, must expand */
			{
				ofs=0;
				for (i=0; i<nrows*scanlinesize; i+=3)	/* one element is 3 bytes, thus the increment */
				{
					if (ofs < h*w-1)
					{
					
						/* 3 bytes in buffer (xx-yy-zz) become 2 shorts (0xxy-0yzz) */
					
						// dp (3, "Src row %5d  Dest offset %5d   line %5d   %2X%2X%2X to %4X-%4X\n",
						// 		row, ofs, i,
						// 		buf[i],buf[i+1],buf[i+2],
						// 		((unsigned short)buf[i] << 4) + (buf[i+1] >> 4),
						// 		(((unsigned short)buf[i+1] << 8) & 0x0f00) + buf[i+2] );
					
						ps[w*row+ofs  ] = ((unsigned short)buf[i] << 4) + (buf[i+1] >> 4);
						ps[w*row+ofs+1] = (((unsigned short)buf[i+1] << 8) & 0x0f00) + buf[i+2] ;
						ofs += 2;
					}
				}
			}
			else if (bpp==1)	/* 1 bit per pixel (fax) - expand */
			{
				ofs=0;
				for (i=0; i<nrows*scanlinesize; i++)
				{
					if (ofs < h*w-1)
					{
					
						/* expand each bit of buf into one byte of p. */

						for (j=0; j<8; j++)
							p[w*row+ofs+j  ] = (buf[i] >> (7-j)) & 0x01;	/* Big-endian version? */
						ofs += 8;
					}
				}
			
			}
			else if ((bpp==16) && (elemsize==6))	/* 48-bit color RGB */
			{
				ofs=0;
				for (i=0; i<nrows*scanlinesize; i+=6)	/* one element is 3 shorts, thus the increment */
				{
					if (ofs < h*w-1)
					{
					
						/* 3 shorts in buffer are (rrr-ggg-bbb) */

						ps[w*row+ofs      ] = ((unsigned short)(buf[i+1]<<8) + buf[i+0]);
						ps[w*row+ofs+  h*w] = ((unsigned short)(buf[i+3]<<8) + buf[i+2]);
						ps[w*row+ofs+2*h*w] = ((unsigned short)(buf[i+5]<<8) + buf[i+4]);
						ofs += 1;
					}
				}
			
			}
			else			/* No expansion, copy verbatim */
			{
				memcpy (p+elemsize*row*w, buf, nrows*scanlinesize);
			}
			
		}
		_TIFFfree(buf);
	}
	else
		return -1;
		
	return 0;
}

void read_tiff(const char *filename, img_t *tiff) {

    TIFF *tiff_handle;
    uint32_t xsize, ysize;
    uint16_t bitspersample, samplesperpixel, planarconfig, photometric, sampleformat;
    
    tiff_handle = TIFFOpen(filename, "r");
    if(!tiff_handle) {
        printf("Unable to open tiff\n\n");
        exit(0x10);
    }

    TIFFGetField (tiff_handle, TIFFTAG_IMAGEWIDTH, &xsize);
	TIFFGetField (tiff_handle, TIFFTAG_IMAGELENGTH, &ysize);
	TIFFGetField (tiff_handle, TIFFTAG_BITSPERSAMPLE, &bitspersample);
	TIFFGetField (tiff_handle, TIFFTAG_SAMPLESPERPIXEL, &samplesperpixel);
	TIFFGetField (tiff_handle, TIFFTAG_PHOTOMETRIC, &photometric);
	TIFFGetField (tiff_handle, TIFFTAG_SAMPLEFORMAT, &sampleformat);

    tiff->xmax = xsize;
    tiff->ymax = ysize;

    tiff->data = (float**)calloc(tiff->xmax * tiff->ymax, 4);


    TIFFGetField(tiff_handle, TIFFTAG_PLANARCONFIG, &planarconfig);
    // printf("\nplanarconfig = %d\n", planarconfig);
    TIFFReadContigStripData(tiff_handle,4,bitspersample,tiff->data);
    TIFFClose(tiff_handle);
}


int write_tiff(const char *fname, img_t *img) {
    
    TIFF *tif;
    char* p;

	tif = TIFFOpen(fname, "w");
	if (!tif) {
        printf("\nCould not open tif during write\n");
        return -1;
    }
	/* Let's start with some general tags */

	TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, img->xmax);
	TIFFSetField(tif, TIFFTAG_IMAGELENGTH, img->ymax);
	TIFFSetField(tif, TIFFTAG_COMPRESSION, 0);

	TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);
	TIFFSetField(tif, TIFFTAG_ORIENTATION, ORIENTATION_TOPLEFT);
	TIFFSetField(tif, TIFFTAG_RESOLUTIONUNIT, (int)2);
	TIFFSetField(tif, TIFFTAG_XRESOLUTION, 1200);
	TIFFSetField(tif, TIFFTAG_YRESOLUTION, 1200);
	TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);

    TIFFSetField(tif, TIFFTAG_SAMPLESPERPIXEL, 1);
    TIFFSetField(tif, TIFFTAG_BITSPERSAMPLE,  32);
    TIFFSetField(tif, TIFFTAG_PHOTOMETRIC,    PHOTOMETRIC_MINISBLACK);
    TIFFSetField(tif, TIFFTAG_SAMPLEFORMAT,   SAMPLEFORMAT_IEEEFP);
    
    p = (char*)img->data;
    TIFFSetField(tif, TIFFTAG_ROWSPERSTRIP, img->ymax);
	TIFFWriteRawStrip(tif, 0, p, 4*img->xmax*img->ymax);

	TIFFClose(tif);

	return 0;
}

void copyimg(img_t *in, img_t *out) {
    out->xmax = in->xmax;
    out->ymax = in->ymax;
    out->data = (float**)calloc(out->xmax*out->ymax, 4);
    memcpy(out->data, in->data, 4*out->xmax*out->ymax);
}


/* Methods to make sinogram */
// Assume parallel beam with 1 degree ingrements //
// default interporlate is bilinear
// z is 1
// arraywidth is 512 for detector arr
void make_sinogram(img_t *img){
    float d, deltad;
    float deltaphi, phi;
    img_t sinogram, tracegram;
    float s1x, s1y, s2x, s2y, sdx, sdy;
    int a,cnt,line,column,xm,ym,ix,iy,sc,nv,aw2;
    uint64_t idx;
    float x1,y1,x2,y2,h,dx,dy,x,y;
    float* p;   
    rgbtriplet* cp; 
    rgbtriplet r1,r2;   
    double buf, atten;  
    double *sintbl, *costbl;    

    deltaphi = 1;
    nv = (int) (0.5+360.0/deltaphi);
    xm = img->xmax; ym = img->ymax;
    aw2 = 256; /* 512/2 */

    allocate_image(&sinogram, ARRWID, nv, FLOATT);
    p = (float*)sinogram.data;
    allocate_image(&tracegram, 2*xm, 2*ym, RGBTRPT);
    cp = (rgbtriplet*)tracegram.data;
    d = 0.5*sqrt( SQR(xm) + SQR(ym));

    deltad = 2*d/ARRWID;
    sintbl = (double*)calloc(nv ,sizeof(double));
    costbl = (double*)calloc(nv ,sizeof(double));
    sc = 0;
    for(phi = 0; phi < 360; phi += deltaphi) { // one revolution
        sintbl[sc] = cos (PHI);
		costbl[sc] = -sin (PHI);
		sc++;
    }

    line=0; idx=0; sc=0;
	for (phi=0; phi<360; phi+=deltaphi)	/* One full revolution including redundancy */
	{
/*		s1x = - (d+2)*costbl[sc]; s1y = - (d+2)*sintbl[sc];	   Source midpoint */
		s1x = - d*costbl[sc]; s1y = - d*sintbl[sc];			/* Source midpoint */
		s2x = -s1x; s2y = -s1y;								/* Detector midpoint */
		column=0;

		for (a = 0; a<ARRWID; a++)				/* One detector line */
		{
			sdx = -deltad*((a-aw2)*sintbl[sc]);
			sdy =  deltad*((a-aw2)*costbl[sc]);			/* Detector element offset */
			x2=s2x+sdx; y2=s2y+sdy;					/* Endpoint of beam */
			x1=s1x+sdx; y1=s1y+sdy;				/* Startpoint at source */
			dx = x2-x1; dy= y2-y1;
			h = sqrt (SQR(dx)+SQR(dy));				/* Path length */
			dx /= h; dy /= h;						/* steps for unit step length */
			
			/* Walk along the beam path */
			
			x=x1; y=y1; cnt=0; atten=0.0;
			do
			{
				buf = ireadbuf(img, x+0.5*xm,y+0.5*ym);
				atten += buf;
				
				/* This part generates the X-ray traces in the secondary wnd */
				if ((x+xm>=0) && (x+xm<2*xm) && (y+ym>=0) && (y+ym<2*ym))
				{
					r2 = (rgbtriplet){0,0,0};
					r2.blue = (int) (10.0+240*phi/360);
					r2.red = 255-r2.blue;
					r1=r2; r1.green=255;
					
					ix = (int) (0.5+x); iy = (int) (0.5+y);

					if ((x+0.5*xm>=0) && (x+0.5*xm<xm) && (y+0.5*ym>=0) && (y+0.5*ym<ym))
						cp[ ( (ix+xm) + 2*xm*(iy+ym) )] = r1;
					else
						cp[ ( (ix+xm) + 2*xm*(iy+ym) )] = r2;
				}

				x+=dx; y+=dy; cnt++;
			}
			while ( (SQR(x-x2)+SQR(y-y2)) > 1.0);	/* Proximity of endpoint */
			p[column + (ARRWID)*line] = atten;
			column++;
		}
		line++;
		sc++;
	}


} // make_sinogram

void allocate_image(img_t *img, int x, int y, int type) {
    img->xmax = x;
    img->ymax = y;

    if(type == 1) { //float
        img->data = (float**)calloc(x*y, 4);    
    } else if(type == 2) { //rgb triplet
        img->data = (float**)calloc(x*y, 3);
    }

    if(img->data == NULL) {
        printf("Could not allocate image\n");
        exit(1);
    }
} // allocate_image

double ireadbuf(img_t *img, double x, double y) {
    double a1,a2,a3,a4,a5,a6;	/* 4 nearest neighbors */
    double xval[4];				/* 4 row interpolations in bi-cubic */
    double A[4];				/* Cubic interpolation coefficents */
    int x1,x2,y1,y2,i;

    x1=floor (x); x2=floor (x+1);	/* Determine the 4 neighbors */
    y1=floor (y); y2=floor (y+1);
    a1 = readbuf_flt (img,x1,y1);
    a2 = readbuf_flt (img,x2,y1);
    a3 = readbuf_flt (img,x1,y2);
    a4 = readbuf_flt (img,x2,y2);
    
    x = x-x1; y = y-y1;	/* Should always be in range 0...1 */
    a5 = (1-x)*a1 + x*a2;
    a6 = (1-x)*a3 + x*a4;
    return (1-y)*a5 + y*a6;
} // ireadbuf

double readbuf_flt (img_t *img, int x, int y) {
    float* p4;

	if (x<0) x=0; else if (x>=img->xmax) x=img->xmax-1;
	if (y<0) y=0; else if (y>=img->ymax) y=img->ymax-1;
 
	p4=(float*)img->data;
    return p4[x+img->xmax*(y+img->ymax)];
	
} // readbuf_flt