#include "hip/hip_runtime.h"
#include "wsart_support.h"

int TIFFReadContigStripData(TIFF* tif, int elemsize, int bpp, void* dataptr) {
    unsigned char *buf;
    tsize_t scanlinesize = TIFFScanlineSize(tif);
    long tiffstripsize;
    unsigned char* p;
    unsigned short* ps;
    int i,j;
    uint32 row, h, w, ofs;
    uint32 rowsperstrip = (uint32)-1;

	p = (unsigned char*)dataptr; ps= (unsigned short*)dataptr;

	tiffstripsize = TIFFStripSize(tif); /* usually scan line (=row) size * rows per strip */
	buf = (unsigned char *)_TIFFmalloc(tiffstripsize);
	if (buf) 
	{

		TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &h);
		TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &w);
		TIFFGetField(tif, TIFFTAG_ROWSPERSTRIP, &rowsperstrip);

		printf("Image length %d, image width (height) %d\n",(int)h,(int)w);
		printf("Scan line size: %d, strip size %li, rows per strip %d\n",
					(int)scanlinesize,tiffstripsize,rowsperstrip);
		if (bpp==12)
		{
			printf("Performing expansion from 12 to 16 bits/sample\n");
		}
		else if (bpp==1)	/* 1 bit per pixel (fax) - expand */
		{
			printf("Performing expansion from 1 to 8 bits/sample\n");
		}
		else if ((bpp==16)  && (elemsize==6))
		{
			printf("Performing conversion from 48-bit RGB to Z stack\n");
		}
		

		ofs=0;
		for (row = 0; row < h; row += rowsperstrip) 
		{
			/* nrow is the number of rows to be read. At the end of the image we have less of them */
			uint32 nrows = (row+rowsperstrip > h ? h-row : rowsperstrip);

			/* Read the next strip, decode compression if necessary */
			tstrip_t strip = TIFFComputeStrip(tif, row, 0);
			if (TIFFReadEncodedStrip(tif, strip, buf, nrows*scanlinesize) < 0) 
			{
				printf("Error reading strip data");
				return -1;
			} 
		
			if (bpp==12)		/* 12 bit per pixel, must expand */
			{
				ofs=0;
				for (i=0; i<nrows*scanlinesize; i+=3)	/* one element is 3 bytes, thus the increment */
				{
					if (ofs < h*w-1)
					{
					
						/* 3 bytes in buffer (xx-yy-zz) become 2 shorts (0xxy-0yzz) */
					
						// dp (3, "Src row %5d  Dest offset %5d   line %5d   %2X%2X%2X to %4X-%4X\n",
						// 		row, ofs, i,
						// 		buf[i],buf[i+1],buf[i+2],
						// 		((unsigned short)buf[i] << 4) + (buf[i+1] >> 4),
						// 		(((unsigned short)buf[i+1] << 8) & 0x0f00) + buf[i+2] );
					
						ps[w*row+ofs  ] = ((unsigned short)buf[i] << 4) + (buf[i+1] >> 4);
						ps[w*row+ofs+1] = (((unsigned short)buf[i+1] << 8) & 0x0f00) + buf[i+2] ;
						ofs += 2;
					}
				}
			}
			else if (bpp==1)	/* 1 bit per pixel (fax) - expand */
			{
				ofs=0;
				for (i=0; i<nrows*scanlinesize; i++)
				{
					if (ofs < h*w-1)
					{
					
						/* expand each bit of buf into one byte of p. */

						for (j=0; j<8; j++)
							p[w*row+ofs+j  ] = (buf[i] >> (7-j)) & 0x01;	/* Big-endian version? */
						ofs += 8;
					}
				}
			
			}
			else if ((bpp==16) && (elemsize==6))	/* 48-bit color RGB */
			{
				ofs=0;
				for (i=0; i<nrows*scanlinesize; i+=6)	/* one element is 3 shorts, thus the increment */
				{
					if (ofs < h*w-1)
					{
					
						/* 3 shorts in buffer are (rrr-ggg-bbb) */

						ps[w*row+ofs      ] = ((unsigned short)(buf[i+1]<<8) + buf[i+0]);
						ps[w*row+ofs+  h*w] = ((unsigned short)(buf[i+3]<<8) + buf[i+2]);
						ps[w*row+ofs+2*h*w] = ((unsigned short)(buf[i+5]<<8) + buf[i+4]);
						ofs += 1;
					}
				}
			
			}
			else			/* No expansion, copy verbatim */
			{
				memcpy (p+elemsize*row*w, buf, nrows*scanlinesize);
			}
			
		}
		_TIFFfree(buf);
	}
	else
		return -1;
		
	return 0;
}

void read_tiff(const char *filename, img_t *tiff) {

    TIFF *tiff_handle;
    uint32_t xsize, ysize;
    uint16_t bitspersample, samplesperpixel, planarconfig, photometric, sampleformat;
    
    tiff_handle = TIFFOpen(filename, "r");
    if(!tiff_handle) {
        printf("Unable to open tiff\n\n");
        exit(0x10);
    }

    TIFFGetField (tiff_handle, TIFFTAG_IMAGEWIDTH, &xsize);
	TIFFGetField (tiff_handle, TIFFTAG_IMAGELENGTH, &ysize);
	TIFFGetField (tiff_handle, TIFFTAG_BITSPERSAMPLE, &bitspersample);
	TIFFGetField (tiff_handle, TIFFTAG_SAMPLESPERPIXEL, &samplesperpixel);
	TIFFGetField (tiff_handle, TIFFTAG_PHOTOMETRIC, &photometric);
	TIFFGetField (tiff_handle, TIFFTAG_SAMPLEFORMAT, &sampleformat);

    tiff->xmax = xsize;
    tiff->ymax = ysize;

    tiff->data = (float**)calloc(tiff->xmax * tiff->ymax, 4);


    TIFFGetField(tiff_handle, TIFFTAG_PLANARCONFIG, &planarconfig);
    // printf("\nplanarconfig = %d\n", planarconfig);
    TIFFReadContigStripData(tiff_handle,4,bitspersample,tiff->data);
    TIFFClose(tiff_handle);
}


int write_tiff(const char *fname, img_t *img) {
    
    TIFF *tif;
    char* p;

	tif = TIFFOpen(fname, "w");
	if (!tif) {
        printf("\nCould not open tif during write\n");
        return -1;
    }
	/* Let's start with some general tags */

	TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, img->xmax);
	TIFFSetField(tif, TIFFTAG_IMAGELENGTH, img->ymax);
	TIFFSetField(tif, TIFFTAG_COMPRESSION, 0);

	TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);
	TIFFSetField(tif, TIFFTAG_ORIENTATION, ORIENTATION_TOPLEFT);
	TIFFSetField(tif, TIFFTAG_RESOLUTIONUNIT, (int)2);
	TIFFSetField(tif, TIFFTAG_XRESOLUTION, 1200);
	TIFFSetField(tif, TIFFTAG_YRESOLUTION, 1200);
	TIFFSetField(tif, TIFFTAG_PLANARCONFIG, PLANARCONFIG_CONTIG);

    TIFFSetField(tif, TIFFTAG_SAMPLESPERPIXEL, 1);
    TIFFSetField(tif, TIFFTAG_BITSPERSAMPLE,  32);
    TIFFSetField(tif, TIFFTAG_PHOTOMETRIC,    PHOTOMETRIC_MINISBLACK);
    TIFFSetField(tif, TIFFTAG_SAMPLEFORMAT,   SAMPLEFORMAT_IEEEFP);
    
    p = (char*)img->data;
    TIFFSetField(tif, TIFFTAG_ROWSPERSTRIP, img->ymax);
	TIFFWriteRawStrip(tif, 0, p, 4*img->xmax*img->ymax);

	TIFFClose(tif);

	return 0;
}

void copyimg(img_t *in, img_t *out) {
    out->xmax = in->xmax;
    out->ymax = in->ymax;
    out->data = (float**)calloc(out->xmax*out->ymax, 4);
    memcpy(out->data, in->data, 4*out->xmax*out->ymax);
}


/* Methods to make sinogram */
// Assume parallel beam with 1 degree ingrements //
// default interporlate is bilinear
// z is 1
// arraywidth is 512 for    detector arr
void make_sinogram(img_t *img){
    float deltad;
    float deltaphi, phi;
    img_t sinogram;
    float s1x,s1y,s2x,s2y;		/* Source and detector midpoints; offset */
    int a,d,t,line,xm,ym,nv,aw2;
    float* p;
    double buf, atten;
    double cphi, sphi;

	/* Get memory for the sinogram image */
	
	deltaphi = 1;					/* Angular increment in degrees */
	nv = 360.0;	/* Number of views */
	xm = img->xmax; ym = img->ymax;		/* we use that all over the place */
	aw2 = ARRWID/2;					/* Midpoint of detector array */
	d = MIN(xm,ym);						/* Diameter of largest circle completely inside box */
	deltad = (float)d/(float)ARRWID;	/* Size of one detector element in image pixels */

	allocate_image (&sinogram, ARRWID, nv, FLOATT);
	p = (float*)sinogram.data;
	
    line=0;
	for (phi=0; phi<360; phi+=deltaphi)	/* One full revolution including redundancy */
	{
		/* We can pre-compute cos phi and sin phi for this angle */

		cphi = COS(phi);
		sphi = SIN(phi);
		/* Now compute one projection (one view) */

		for (a=0; a<ARRWID; a++)		/* Run along the detector array */
		{
			atten=0;						/* Cumulative attenuation along ray */
			for (t=0; t<d; t++)				/* Run along one line */
			{
				/* convert a and t into x,y of an unrotated image coordinate system */

				s1x = t-d/2;				/* horizontal offset from image center */
				s1y = (a-aw2)*deltad;		/* Vertical offset of ray from image center */

				/* Rotate s1x,s1y into the image coordinate system */

				s2x = xm/2 + s1x*sphi - s1y*cphi;
				s2y = ym/2 -s1x*cphi - s1y*sphi;

				/* add image value at that point to total attenuation */

				if ((s2x>=0) && (s2x<xm) && (s2y>=0) && (s2y<ym))
				{
					buf = ireadbuf (img, s2x, s2y);
					atten += buf;
				}
                //printf("phi is %f, a is %d, t is %d, atten is %f\n", phi, a, t, atten);

			}
            //printf("phi is %f, a is %d, t is %d, atten is %f\n", phi, a, t, atten);

			/* Place attenuation in sinogram buffer */
            p[a + (line*sinogram.xmax)] = atten;
		}
		line++;
	}

	handback (img, &sinogram);
	//if (!macrorun) reset_progress ();

} // make_sinogram

void allocate_image(img_t *img, int x, int y, int type) {
    img->xmax = x;
    img->ymax = y;
    printf("got here and x and y is %d %d\n", x, y);
    if(type == 1) { //float
        img->data = (float**)calloc(x*y, 4);    
    } else if(type == 2) { //rgb triplet
        //img->data = (float**)calloc(x*y, 3);
    }
    //img->data = (float**)calloc(x*y, 4);

    if(img->data == NULL) {
        printf("Could not allocate image\n");
        exit(1);
    }
} // allocate_image

double ireadbuf(img_t *img, double x, double y) {
    //double a1,a2,a3,a4,a5,a6;	/* 4 nearest neighbors */
    int x1;
    int y1;

    // x1=floor(x); x2=floor(x+1);	/* Determine the 4 neighbors */
    // y1=floor(y); y2=floor(y+1);
    // a1 = readbuf_flt (img,x1,y1);
    // a2 = readbuf_flt (img,x2,y1);
    // a3 = readbuf_flt (img,x1,y2);
    // a4 = readbuf_flt (img,x2,y2);
    x1 = floor (0.5+x);
    y1 = floor (0.5+y);
    printf("x1 is %d and y1 is %d\n", x1, y1);
    // exit(0);
    return readbuf_flt (img, x1,y1);
    
    // x = x-x1; y = y-y1;	/* Should always be in range 0...1 */
    // a5 = (1-x)*a1 + x*a2;
    // a6 = (1-x)*a3 + x*a4;
    // return (1-y)*a5 + y*a6;
} // ireadbuf

double readbuf_flt (img_t *img, int x, int y) {
    float* p4;

	if (x<0) x=0; else if (x>=img->xmax) x=img->xmax-1;
	if (y<0) y=0; else if (y>=img->ymax) y=img->ymax-1;
 
	p4=(float*)img->data;
    float temp = p4[x+img->xmax*(y+img->ymax)];
    printf("Temp is %f\n", temp);
    return p4[x+img->xmax*(y+img->ymax)];
	
} // readbuf_flt

void handback(img_t *dest, img_t *src) {
	freebuf (dest);
	dest->data = src->data; 
	dest->xmax = src->xmax;
	dest->ymax = src->ymax;
	//make_minmax (dest);
}

void freebuf(img_t *img) {
    if (img->data) free(img->data);
	img->data = NULL;
	img->xmax = img->ymax = 0;
}