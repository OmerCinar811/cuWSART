#include "stdio.h"
#include "stdlib.h"
#include "wsart_support.h"
// #include "png.h"

/*
    Program flow                                ----------
                                                V        |
    Read image --> make sinogram --> wsart --> wbp --> wfp --> wbp --> out

        A
        |
        |
   On this step

*/

int main(int argc, char const *argv[]) {
    
    const char *in_file_name;
    //const char *sin_file_name = "sinogram.png";
    const char *out_file_name;
    
    hipError_t cuda_ret = hipErrorUnknown;
    //printf("got here");
   // exit(0);
    
    if(argc == 1) {
        printf("\nERROR: Missing Input file and output file arguments\n\n");
        printf("\tExpected 2 arguments, received 0\n\n");
        return 1;
    } else if(argc == 2) {
        printf("\nERROR: Missing Input file or output file argument\n\n");
        printf("\tExpected 2 arguments, received 1\n\n");
        return 2;
    } else if(argc == 3) {
        in_file_name = argv[1];
        out_file_name = argv[2];
    } else {
        printf("\n    Invalid input parameters!"
               "\n    Usage: ./wsart <input_file> <output_file>"
               "\n\n");
        exit(0);
    }

    img_t inimg;
    img_t outimg;
    //printf("got here");
    //exit(0);
    read_tiff(in_file_name, &inimg);

    copyimg(&inimg, &outimg);

    write_tiff(out_file_name, &outimg);

    return 0;
}
