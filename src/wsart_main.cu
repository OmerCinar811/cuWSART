#include "stdio.h"
#include "stdlib.h"
#include "wsart_support.h"
// #include "png.h"

/*
    Program flow                                ----------
                                                V        |
    Read image --> make sinogram --> wsart --> wbp --> wfp --> wbp --> out

        A
        |
        |
   On this step

*/

int main(int argc, char const *argv[]) {
    

    FILE *in_file;
    //FILE *sin_file;
    FILE *out_file;
    const char *in_file_name;
    //const char *sin_file_name = "sinogram.png";
    const char *out_file_name;
    png_t in_png;
    //png_t sin_png;
    png_t out_png;

    //hipError_t cuda_ret;
    
    if(argc == 1) {
        printf("\nERROR: Missing Input file and output file arguments\n\n");
        printf("\tExpected 2 arguments, received 0\n\n");
        return 1;
    } else if(argc == 2) {
        printf("\nERROR: Missing Input file or output file argument\n\n");
        printf("\tExpected 2 arguments, received 1\n\n");
        return 2;
    } else if(argc == 3) {
        in_file_name = argv[1];
        out_file_name = argv[2];
    } else {
        printf("\n    Invalid input parameters!"
               "\n    Usage: ./wsart <input_file> <output_file>"
               "\n\n");
        exit(0);
    }

    in_file = fopen(in_file_name, "rb");
    out_file = fopen(out_file_name, "wb");
    //fopen sinfile

    // read_png(in_file, &in_png);
    // memcpy(&out_png, &in_png, sizeof(in_png));
    // write_png(out_file, &out_png);
 

    fclose(in_file);
    fclose(out_file);
    //fclose sinfile

    return 0;
}
