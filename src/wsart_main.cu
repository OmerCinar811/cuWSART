#include "stdio.h"
#include "stdlib.h"
#include "wsart_support.h"
// #include "png.h"

/*
    Program flow                                ----------
                                                V        |
    Read image --> make sinogram --> wsart --> wbp --> wfp --> wbp --> out

*/

int main(int argc, char const *argv[]) {
    

    FILE *in_file;
    FILE *sin_file;
    FILE *out_file;
    const char *in_file_name;
    const char *sin_file_name = "sinogram.png";
    const char *out_file_name;

    hipError_t cuda_ret;

    //fopen infile
    //fopen sinfile
    //fopen outfile

    if(argc == 1) {
        printf("\nERROR: Missing Input file and output file arguments\n");
        printf("Expected 2 arguments, received 0");
        return 1;
    } else if(argc == 2) {
        printf("\nERROR: Missing Input file or output file argument\n");
        printf("Expected 2 arguments, received 1");
        return 2;
    } else if(argc == 3) {
        in_file_name = argv[1];
        out_file_name = argv[2];
    } else {
        printf("\n    Invalid input parameters!"
               "\n    Usage: ./wsart <input_file> <output_file>"
               "\n");
        exit(0);
    }


    //fclose infile
    //fclose sinfile
    //fclose outfile

    return 0;
}
