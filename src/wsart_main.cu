#include "stdio.h"
#include "stdlib.h"
#include "wsart_support.h"
// #include "png.h"

/*
    Program flow                                ----------
                                                V        |
    Read image --> make sinogram --> wsart --> wbp --> wfp --> wbp --> out

        A
        |
        |
   On this step

*/

int main(int argc, char const *argv[]) {
    

    FILE *in_file;
    FILE *sin_file;
    FILE *out_file;
    const char *in_file_name;
    const char *sin_file_name = "sinogram.png";
    const char *out_file_name;
    png_t *in_png;
    png_t *sin_png;
    png_t *out_png;

    hipError_t cuda_ret;
    if(argc == 1) {
        printf("\nERROR: Missing Input file and output file arguments\n");
        printf("Expected 2 arguments, received 0");
        return 1;
    } else if(argc == 2) {
        printf("\nERROR: Missing Input file or output file argument\n");
        printf("Expected 2 arguments, received 1");
        return 2;
    } else if(argc == 3) {
        in_file_name = argv[1];
        out_file_name = argv[2];
    } else {
        printf("\n    Invalid input parameters!"
               "\n    Usage: ./wsart <input_file> <output_file>"
               "\n");
        exit(0);
    }

    in_file = fopen(in_file_name, "rb");
    //fopen sinfile
    //fopen outfile


    fclose(in_file);
    //fclose sinfile
    //fclose outfile

    return 0;
}
